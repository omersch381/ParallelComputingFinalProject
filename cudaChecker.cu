#include "hip/hip_runtime.h"
#include <hip/hip_runtime.h>
#include <hip/hip_runtime_api.h>
#include <stdlib.h>
#include <string.h>
extern "C" {
  #include "cudaChecker.h"
}
#define GROUP_STRING_SIZE_LIMIT 7

// Declarations

__global__ void
areTheCharsInGroup(char mainChar, char checkedChar,
                   char *groupToCheck,
                   int arraySize, int *areThey);

// Implementations

__global__ void
areTheCharsInGroup(char mainChar, char checkedChar,
                   char* groupToCheck,
                   int arraySize, int *areThey) {
  int isMainCharInTheGroup = 0;
  int isCheckedCharInTheGroup = 0;

  for (int i = 0; i < arraySize; i++) {
    for (int j = 0; j < GROUP_STRING_SIZE_LIMIT; j++) {
      if (groupToCheck[i * GROUP_STRING_SIZE_LIMIT + j]) {
        if (mainChar == groupToCheck[i * GROUP_STRING_SIZE_LIMIT + j])
          isMainCharInTheGroup = 1;
        if (checkedChar == groupToCheck[i * GROUP_STRING_SIZE_LIMIT + j])
          isCheckedCharInTheGroup = 1;
      }
    }
  }
  if (isMainCharInTheGroup && isCheckedCharInTheGroup)
    *areThey = 1;
  else {
    *areThey = 0;
    isMainCharInTheGroup = 0;
    isCheckedCharInTheGroup = 0;
  }
}

extern "C" int areTheCharsInGroupGPU(char mainChar, char checkedChar,
                          char groupToCheck[][GROUP_STRING_SIZE_LIMIT],
                          int arraySize) {
  // Error code to check return values for CUDA calls
  hipError_t err = hipSuccess;

  size_t groupToCheckSize = arraySize * GROUP_STRING_SIZE_LIMIT * sizeof(char);

  // Allocate memory on GPU to copy the mainSequence from the host
  // char groupToCheckDevicePointer[][GROUP_STRING_SIZE_LIMIT] = {};
  char* groupToCheckDevicePointer = 0;

  err = hipMalloc((void **)&groupToCheckDevicePointer, groupToCheckSize);
  if (err != hipSuccess) {
    fprintf(stderr, "Failed to allocate device memory - %s\n",
            hipGetErrorString(err));
    exit(EXIT_FAILURE);
  }

  char* tester = (char*) malloc(groupToCheckSize);
  memcpy(tester,groupToCheck,groupToCheckSize);
  // Copy mainSequence from host to the GPU memory
  err = hipMemcpy(groupToCheckDevicePointer, tester, groupToCheckSize,
    hipMemcpyHostToDevice);


  // // Copy mainSequence from host to the GPU memory
  // err = hipMemcpy(groupToCheckDevicePointer, groupToCheck, groupToCheckSize,
  //                  hipMemcpyHostToDevice);
  if (err != hipSuccess) {
    fprintf(stderr, "Failed to copy data from host to device - %s\n",
            hipGetErrorString(err));
    exit(EXIT_FAILURE);
  }

  // Are they in the same group
  int *areThey = (int *)malloc(sizeof(int));
  *areThey = 0;

  // Launch the Kernel
  int threadsPerBlock = 64;
  int blocksPerGrid =
      (groupToCheckSize + threadsPerBlock - 1) / threadsPerBlock;
  areTheCharsInGroup<<<blocksPerGrid, threadsPerBlock>>>(
      mainChar, checkedChar, groupToCheckDevicePointer, arraySize, areThey);
  err = hipGetLastError();
  if (err != hipSuccess) {
    fprintf(stderr, "Failed to launch vectorAdd kernel -  %s\n",
            hipGetErrorString(err));
    exit(EXIT_FAILURE);
  }

  // // Copy the  result from GPU to the host memory.
  // err = hipMemcpy(groupToCheck, groupToCheckDevicePointer, groupToCheckSize,
  //                  hipMemcpyDeviceToHost);
  // if (err != hipSuccess) {
  //   fprintf(stderr, "Failed to copy result array from device to host -%s\n",
  //           hipGetErrorString(err));
  //   exit(EXIT_FAILURE);
  // }

  // Free allocated memory on GPU - mainSequenceDevicePointer

  // hipFree(groupToCheckDevicePointer);

  if (hipFree(groupToCheckDevicePointer) != hipSuccess) {
    fprintf(stderr, "Failed to free device data - %s\n",
            hipGetErrorString(err));
    exit(EXIT_FAILURE);
  }

  free(tester);

  return *areThey;
}
