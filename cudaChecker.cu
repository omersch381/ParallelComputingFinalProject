#include "hip/hip_runtime.h"
#include <hip/hip_runtime.h>
#include <hip/hip_runtime_api.h>
#include <stdlib.h>
#include <string.h>
#define GROUP_STRING_SIZE_LIMIT 7

// Declarations

int areTheCharsInGroupGPU(char mainChar, char checkedChar,
                          const char groupToCheck[][GROUP_STRING_SIZE_LIMIT],
                          int arraySize);

__global__ void
areTheCharsInGroup(char mainChar, char checkedChar,
                   const char groupToCheck[][GROUP_STRING_SIZE_LIMIT],
                   int arraySize, int *areThey);

// Implementations

__global__ void
areTheCharsInGroup(char mainChar, char checkedChar,
                   const char groupToCheck[][GROUP_STRING_SIZE_LIMIT],
                   int arraySize, int *areThey) {
  int isMainCharInTheGroup = 0;
  int isCheckedCharInTheGroup = 0;

  for (int i = 0; i < arraySize; i++) {
    for (int j = 0; j < GROUP_STRING_SIZE_LIMIT; j++) {
      if (groupToCheck[i][j]) {
        if (mainChar == groupToCheck[i][j])
          isMainCharInTheGroup = 1;
        if (checkedChar == groupToCheck[i][j])
          isCheckedCharInTheGroup = 1;
      }
    }
  }
  if (isMainCharInTheGroup && isCheckedCharInTheGroup)
    *areThey = 1;
  else {
    *areThey = 0;
    isMainCharInTheGroup = 0;
    isCheckedCharInTheGroup = 0;
  }
}

int areTheCharsInGroupGPU(char mainChar, char checkedChar,
                          const char groupToCheck[][GROUP_STRING_SIZE_LIMIT],
                          int arraySize) {
  // Error code to check return values for CUDA calls
  hipError_t err = hipSuccess;

  size_t groupToCheckSize = arraySize * GROUP_STRING_SIZE_LIMIT * sizeof(char);

  // Allocate memory on GPU to copy the mainSequence from the host
  int *groupToCheckDevicePointer;

  err = hipMalloc((void **)&groupToCheckDevicePointer, groupToCheckSize);
  if (err != hipSuccess) {
    fprintf(stderr, "Failed to allocate device memory - %s\n",
            hipGetErrorString(err));
    exit(EXIT_FAILURE);
  }

  // Copy mainSequence from host to the GPU memory
  err = hipMemcpy(groupToCheckDevicePointer, groupToCheck, groupToCheckSize,
                   hipMemcpyHostToDevice);
  if (err != hipSuccess) {
    fprintf(stderr, "Failed to copy data from host to device - %s\n",
            hipGetErrorString(err));
    exit(EXIT_FAILURE);
  }

  // Are they in the same group
  int *areThey = (int *)malloc(sizeof(int));
  *areThey = 0;

  // Launch the Kernel
  int threadsPerBlock = 64;
  int blocksPerGrid =
      (groupToCheckSize + threadsPerBlock - 1) / threadsPerBlock;
  areTheCharsInGroup<<<blocksPerGrid, threadsPerBlock>>>(
      mainChar, checkedChar, groupToCheckDevicePointer, arraySize, areThey);
  err = hipGetLastError();
  if (err != hipSuccess) {
    fprintf(stderr, "Failed to launch vectorAdd kernel -  %s\n",
            hipGetErrorString(err));
    exit(EXIT_FAILURE);
  }

  // // Copy the  result from GPU to the host memory.
  // err = hipMemcpy(groupToCheck, groupToCheckDevicePointer, groupToCheckSize,
  //                  hipMemcpyDeviceToHost);
  // if (err != hipSuccess) {
  //   fprintf(stderr, "Failed to copy result array from device to host -%s\n",
  //           hipGetErrorString(err));
  //   exit(EXIT_FAILURE);
  // }

  // Free allocated memory on GPU - mainSequenceDevicePointer
  if (hipFree(groupToCheckDevicePointer) != hipSuccess) {
    fprintf(stderr, "Failed to free device data - %s\n",
            hipGetErrorString(err));
    exit(EXIT_FAILURE);
  }

  return *areThey;
}
